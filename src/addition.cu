/*
This is a simple CUDA program which performs addition using GPU (kernel function) 
*/

#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void add(int a, int b, int *d_c)
{
    *d_c = a + b;
}
int main()
{
    int a,b,c;
    int *d_c;
    a=3;
    b=4;
    // Allocating memory for device pointer of integer
    hipMalloc((void**)&d_c, sizeof(int));
    // Calling Kernel function
    add<<<1,1>>>(a,b,d_c);
    // Copying the result from device to host
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d + %d is %d\n", a, b, c);
    // Free the device pointer
    hipFree(d_c);
    return 0;
}
