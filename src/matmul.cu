//Matrix Multiplication using CUDA


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define TILE_SIZE 16

//matrix multiplication kernel function 
__global__ void MatrixMultiplication( float *device_array1 , float *device_array2 , float *device_result_array , const int SIZE )
{

    __shared__ float Mds [TILE_SIZE][TILE_SIZE] ;
    __shared__ float Nds [TILE_SIZE][TILE_SIZE] ;

    int col = blockDim.x*blockIdx.x + threadIdx.x ;
    int row = blockDim.y*blockIdx.y + threadIdx.y ;
    

    for (int m = 0 ; m<SIZE/TILE_SIZE ; m++ ) 
    {
        Mds[threadIdx.y][threadIdx.x] =  device_array1[row*SIZE + (m*TILE_SIZE + threadIdx.x)]  ;
        Nds[threadIdx.y][threadIdx.x] =  device_array2[(m*TILE_SIZE + threadIdx.y) * SIZE + col] ;
         __syncthreads(); 
        for ( int k = 0; k<TILE_SIZE ; k++ )
            device_result_array[row*SIZE + col]+= Mds[threadIdx.x][k] * Nds[k][threadIdx.y] ;
            __syncthreads();

    }
}
int main ()
{
    const int SIZE = 512 ; 
    int i,j;
    // Host Vairables
    float host_array1[SIZE][SIZE], host_array2[SIZE][SIZE], host_result_array[SIZE][SIZE];
    // Device Variables
    float *device_array1 , *device_array2  ,*device_result_array ; 

    // Inserting values into Arrays
    for (i = 0 ; i<SIZE ; i++ )
    {
        for (j = 0 ; j<SIZE ; j++ )
        {
            host_array1[i][j] = 2;
            host_array2[i][j] = 1 ;
        }
    }

    // Allocate memory for GPU 
    hipMalloc((void **) &device_array1 , SIZE*SIZE*sizeof (int)) ;
    hipMalloc((void **) &device_array2 , SIZE*SIZE*sizeof (int)) ;
    hipMalloc((void **) &device_result_array , SIZE*SIZE*sizeof (int)) ;
 
    // Copying yhe array from Host to Device Array
    hipMemcpy ( device_array1 , host_array1 , SIZE*SIZE*sizeof (int) , hipMemcpyHostToDevice ) ;
    hipMemcpy ( device_array2 , host_array2 , SIZE*SIZE*sizeof (int) , hipMemcpyHostToDevice ) ;
 
    //calling kernal
    dim3 dimGrid ( SIZE/TILE_SIZE , SIZE/TILE_SIZE ) ;
    dim3 dimBlock( TILE_SIZE, TILE_SIZE ) ;
    MatrixMultiplication<<<dimGrid,dimBlock>>> ( device_array1 , device_array2 ,device_result_array , SIZE) ;

    // all gpu function blocked till kernel is working
    //copy back result_array_d to result_array_h
    hipMemcpy(host_result_array , device_result_array , SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost) ;

    //printf the result array
    for ( i = 0 ; i<SIZE ; i++ )
    {
        for ( j = 0 ; j < SIZE ; j++ )
        {
            printf ("%.0f ",host_result_array[i][j] ) ;
        }
    printf ("\n") ;
    }
}
