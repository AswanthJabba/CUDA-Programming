
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void print_from_gpu(void) {
 printf("Hello World! from thread [%d,%d] From device\n", threadIdx.x,blockIdx.x);
}

int main(void) {
 printf("Hello World from host!\n");
 print_from_gpu<<<10,2>>>(); // first parameter is number of Blocks, second parameter is number of threads 
 hipDeviceSynchronize();
 return 0;
}
