/*
This program performs sqaure of a number from 1 to N parallely using N threads and 1 Block
Warning : It is not advisable to practice using single block multiple threads, Please have a look at square2.cu for a better practice of blocks and threads
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 128

__global__ void f(int *dev_a) {
    unsigned int tid = threadIdx.x;

    if(tid < N) {
        dev_a[tid] = tid * tid;
    }
}

int main(void) {

    int host_a[N];
    int *dev_a;
    //Memory allocation for device (GPU)
    hipMalloc((void**)&dev_a, N * sizeof(int));
    for(int i = 0 ; i < N ; i++) {
        host_a[i] = i;
    }
    for(int i = 0 ; i < N ; i++) {
        printf("%d\t ", host_a[i]);
    }
    printf("\n");
    //Copy data from Host(CPU) to device(GPU)
    hipMemcpy(dev_a, host_a, N * sizeof(int), hipMemcpyHostToDevice);
    //Calling GPU kernel (GPu function)
    f<<<1, N>>>(dev_a);
    //Copying back the result from Device(GPU) to Host(CPU)
    hipMemcpy(host_a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0 ; i < N ; i++) {
        printf("%d\t ", host_a[i]);
    }
}
